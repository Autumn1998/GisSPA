#include "hip/hip_runtime.h"
#include "GPU_func.cuh"

__global__ void UpdateSigma(hipfftComplex *d_templates,float *d_buf)
{
	extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    int tid = threadIdx.x;
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = d_templates[i].x;
	sdata[tid+blockDim.x] = d_templates[i].x*d_templates[i].x;
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if(tid<s)
		{
			//sum of data[i] & data[i]^2
			sdata[tid] += sdata[tid + s]; 
			sdata[tid+blockDim.x] += sdata[tid + blockDim.x + s];
		}
		__syncthreads();
	}
	if(tid==0){
		d_buf[blockIdx.x*2] = sdata[0];
		d_buf[blockIdx.x*2+1] = sdata[blockDim.x];
	}
}

__global__  void  generate_mask(int l,hipfftComplex *mask,float r,float *res,float up,float low)
{
	extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    int tid = threadIdx.x;
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	int image_size = l*l;
	int local_id = i % image_size;
    int x = local_id % l;
    int y = local_id / l;

	//Dis^2 between (x,y) and center (l/2,l/2)
	float rr = (x-l/2)*(x-l/2) + (y-l/2)*(y-l/2);
	if(rr>=low && rr<=up)
	{
		mask[i].x = 1;
		mask[i].y = 0;
	}

	//reduction for number of non-zero digits
	sdata[tid] = mask[i].x;
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if(tid<s)	sdata[tid] += sdata[tid+s];
		__syncthreads();
	}
	if(tid==0) res[blockIdx.x] = sdata[0];

}

__global__  void  multiCount_dot(int l,hipfftComplex *mask,hipfftComplex *d_templates,float *constants,float *res)
{
	extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    int tid = threadIdx.x;
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	int image_size = l*l;
	int image_id = i/image_size;

	//Multi constant 1/non-zeros
	if(constants[image_id] != 0) mask[i].x *= 1.0/constants[image_id];

	//reduction for dot
	sdata[tid] = mask[i].x * d_templates[i].x;
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if(tid<s)	sdata[tid] += sdata[tid+s];
		__syncthreads();
	}
	//use res to store dot
	if(tid==0) res[blockIdx.x] = sdata[0];

}

__global__  void  scale_each(int l,hipfftComplex *d_templates,float *ems,double *d_sigmas)
{
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	int image_size = l*l;
	int image_id = i/image_size;

	if(d_sigmas[image_id]-0 < EPS && d_sigmas[image_id]-0 >-EPS ) return;
	d_templates[i].x = (d_templates[i].x - ems[image_id])/d_sigmas[image_id];
}

//mode = 0 (default)  for template
//mode = 1 for raw image
__global__  void  SQRSum_by_circle(hipfftComplex *data, float *ra, float *rb, int nx, int ny, int mode)
{
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = nx*ny;
	if(mode == 1 && i>=image_size) return;
    int local_id = i % image_size;
    int x = local_id % nx;
    int y = local_id / nx;

    float tmp;
	// ri2ap
	tmp=hypotf(data[i].x, data[i].y);
	if (data[i].x==0 && data[i].y==0) 
        data[i].y=0;
	else data[i].y=atan2(data[i].y,data[i].x);
	data[i].x=tmp;

	if(x>nx/2) return;

    //calculate the number of point with fixed distance ('r') from center 
	int r = floor( hypotf(min(y,ny-y) ,min(x,nx-x)) + 0.5) - 1;

	if (r < max(nx,ny)/2 && r >= 0) {
		//Add offset
		r+= RA_SIZE*(i/image_size);
		atomicAdd(&ra[r],data[i].x*data[i].x);
		atomicAdd(&rb[r],1.0);
	}

}

__global__  void  whiten_Tmp(hipfftComplex *data, float *ra, float *rb, int l)
{
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = l*l;  
    int local_id = i % image_size;
    int x = local_id % l;
    int y = local_id / l;
	int r = floor( hypotf(min(y,l-y) ,min(x,l-x)) + 0.5) - 1;

	if (r < l/2 && r >= 0) {
		//Add offset
		r+= RA_SIZE*(i/image_size);
		float fb_infile=ra[r]/rb[r];
		data[i].x=data[i].x/(float)sqrt(fb_infile);
	}

	//ap2ri
	float tmp=data[i].x*sinf(data[i].y);
	data[i].x=data[i].x*cosf(data[i].y);
	data[i].y=tmp;

}


__global__  void  whiten_filetr_weight_Img(hipfftComplex *data, float *ra, float *rb, int nx, int ny, Parameters para)
{
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = nx*ny;  
	if(i>=image_size) return;
    int local_id = i % image_size;
    int x = local_id % nx;
    int y = local_id / nx;

	float rf = hypotf(min(y,ny-y) ,min(x,nx-x));
	int r = floor( rf + 0.5) - 1;
	float ss=rf*para.ds;
	int l = max(nx,ny);

	float v,signal,Ncurve;
    //apply weighting function
	if( r < l/2 && r >= 0){
		v=CTF_AST(x,(y+ny/2)%ny,nx,ny,para.apix,para.ds,para.dfu,para.dfv,para.dfdiff,para.dfang,para.lambda,para.cs,para.ampconst,2);
		signal=(exp(para.bfactor*ss*ss+para.bfactor2*ss+para.bfactor3));
		Ncurve=exp(para.a*ss*ss+para.b*ss+para.b2);

		data[i].x=data[i].x*sqrt((signal*v*v+Ncurve)/signal)/sqrt(ra[r]/rb[r]);
		if(r>(l*para.apix/6)) data[i].x=data[i].x*exp(-100*ss*ss);
	}

	// low pass
	if (r<l*para.apix/para.highres && r >= l*para.apix/para.lowres) {}
	else if(r>=l*para.apix/para.highres && r<l*para.apix/para.highres+8){
		data[i].x=data[i].x*(0.5*cosf(PI*(r-l*para.apix/para.highres)/(2*8))+0.5);
	}
	else if(r>=(l*para.apix/para.lowres-8) && r<l*para.apix/para.lowres && r>=0){
		data[i].x=data[i].x*(0.5*cosf(PI*(l*para.apix/para.lowres-r)/(2*8))+0.5);
	}
	else
		data[i].x=0;

    //apply weighting function
	if( r < l/2 && r >= 0){
		signal/=(para.kk+1);
		Ncurve/=signal;
		data[i].x=data[i].x*sqrt(1/(Ncurve+para.kk*v*v ));
	}
}

__global__ void normalize_Img(hipfftComplex *data,int nx, int ny,float mean)
{
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= nx*ny) return;

	if(mean!=0)	data[i].x=data[i].x/mean;

	//ap2ri
	float tmp=data[i].x*sinf(data[i].y);
	data[i].x=data[i].x*cosf(data[i].y);
	data[i].y=tmp;
}

__global__ void apply_mask(hipfftComplex *data,float d_m,float edge_half_width,int l)
{	
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = l*l;
    int local_id = i % image_size;
    int x = local_id % l;
    int y = local_id / l;

    float r=hypotf(x-l/2,y-l/2);
	if( r > (d_m/2+2*edge_half_width)){
			data[i].x=0;
	}else if (r >= d_m/2){
			float d=0.5*cosf(PI*(r-d_m/2)/(2*edge_half_width))+0.5;
			data[i].x *=d;
	}

}

__global__ void apply_weighting_function(hipfftComplex *data,Parameters para)
{
    int l = para.padding_size;
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = l*l;
    int local_id = i % image_size;
    int x = local_id % l;
    int y = local_id / l;

    float tmp;
	//ri2ap
	tmp=hypotf(data[i].x, data[i].y);
	if (data[i].x==0 && data[i].y==0) 
        data[i].y=0;
	else data[i].y=atan2(data[i].y,data[i].x);
	data[i].x=tmp;

    // low pass
	float r = hypotf(min(y,l-y) ,min(x,l-x) );
	int  r_round = floor(r + 0.5) - 1;
	if (r_round<l*para.apix/para.highres && r_round >= 0) {}
	else if(r_round>=l*para.apix/para.highres && r_round<l*para.apix/para.highres+8){
		data[i].x=data[i].x*(0.5*cosf(PI*(r_round-l*para.apix/para.highres)/(2*8))+0.5);
	}
	else if(r_round>=(l*para.apix/para.lowres-8) && r_round<l*para.apix/para.lowres && r_round>=0){
		data[i].x=data[i].x*(0.5*cosf(PI*(l*para.apix/para.lowres-r_round)/(2*8))+0.5);
	}
	else
		data[i].x=0;
	float ss=r*para.ds;

	float v,signal,Ncurve;
    //apply weighting function
	if( r_round < l/2 && r_round >= 0){
		v=CTF_AST(x,(y+l/2)%l,l,l,para.apix,para.ds,para.dfu,para.dfv,para.dfdiff,para.dfang,para.lambda,para.cs,para.ampconst,2);
		signal=(exp(para.bfactor*ss*ss+para.bfactor2*ss+para.bfactor3))/(para.kk+1);
		Ncurve=exp(para.a*ss*ss+para.b*ss+para.b2)/signal;
		//euler_w[x]=1.68118*ss;
		data[i].x=data[i].x*v*sqrt(1/(Ncurve+para.kk*v*v ));
	}
	
}

__device__ float CTF_AST (int x1, int y1,int nx, int ny, float apix, float ds, float dfu, float dfv, float dfdiff, float dfang ,float lambda, float cs, float ampconst, int mode){
	float v,ss,ag,gamma,df_ast;
	if(x1 > nx/2) 
	{
		x1 = nx-x1;
		y1 = ny-y1;
	}
	ss = ( x1*x1/(float)(nx*nx) + y1*y1/(float)(ny*ny) )/ (apix*apix);
	//ss = hypotf((float)x1,(float)y1-ny/2)*ds*hypotf((float)x1,(float)y1-ny/2)*ds;
	ag=atan2(float(y1-ny/2),float(x1));
	df_ast=0.5*(dfu+dfv+2*dfdiff*cosf(2*(dfang*PI/180-ag)));
	gamma=-2*PI*(cs*2.5e6*lambda*lambda*lambda*ss*ss+df_ast*5000.0*lambda*ss);
	if (mode==0){
		v=(sqrtf(1.0-ampconst*ampconst)*sinf(gamma)+ampconst*cosf(gamma))>0?1.0:-1.0;		//do flipphase
	}
	else if (mode==2){
		v=fabs(sqrtf(1.0-ampconst*ampconst)*sinf(gamma)+ampconst*cosf(gamma));		//	return abs ctf value
	}
	else{
		v=(sqrtf(1.0-ampconst*ampconst)*sinf(gamma)+ampconst*cosf(gamma));		//	return ctf value
	}
	
	return v;
}

//mode = 0 (default)  for template
//mode = 1 for raw image
__global__ void compute_area_sum_ofSQR(hipfftComplex *data,float *res,int nx, int ny, int mode)
{
	extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    int tid = threadIdx.x;
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = nx*ny;
    int local_id = i % image_size;
    int x = local_id % nx;
    int y = local_id / nx;
	int r = floor( hypotf(min(y,ny-y) ,min(x,nx-x)) + 0.5) - 1;
	int l = max(nx,ny);

	if (r < l/2 && r >= 0 && x<=l/2 && mode ==0) {
		sdata[tid] = data[i].x*data[i].x;
		sdata[tid+blockDim.x] = 1;
	}
	else if(mode == 1 && i<nx*ny)
	{
		sdata[tid] = data[i].x*data[i].x;
		sdata[tid+blockDim.x] = 1;
	}else
	{
		sdata[tid]=0;
		sdata[tid+blockDim.x] = 0;
	}
	__syncthreads();
	
	if (tid < 512) { sdata[tid] += sdata[tid + 512]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 512];} __syncthreads();
	if (tid < 256) { sdata[tid] += sdata[tid + 256]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 256];} __syncthreads();
	if (tid < 128) { sdata[tid] += sdata[tid + 128]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 128];} __syncthreads();
	if (tid < 64) { sdata[tid] += sdata[tid + 64]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 64];} __syncthreads();

	if(tid < 32)
	{
		sdata[tid] += sdata[tid + 32]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 32];
		sdata[tid] += sdata[tid + 16]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 16];
		sdata[tid] += sdata[tid + 8]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 8];
		sdata[tid] += sdata[tid + 4]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 4];
		sdata[tid] += sdata[tid + 2]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 2];
		sdata[tid] += sdata[tid + 1]; sdata[tid+blockDim.x] += sdata[tid +blockDim.x+ 1];
	}

	// write result for this block 
	if (tid == 0) {
		res[2*blockIdx.x] = sdata[0];
		res[2*blockIdx.x+1] = sdata[blockDim.x];
	}
}

__global__ void normalize(hipfftComplex *data,int nx, int ny,float *means)
{
    // i <==> global ID
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
    int image_size = nx*ny;
	int template_id = i / image_size;

	if(means[template_id]!=0)	data[i].x=data[i].x/means[template_id];

	//ap2ri
	float tmp=data[i].x*sinf(data[i].y);
	data[i].x=data[i].x*cosf(data[i].y);
	data[i].y=tmp;
}

__global__ void rotate_IMG(float *d_image,float *d_rotated_image,float e,int nx,int ny)
{
	float cose=cos(e*PI/180);
	float sine=sin(e*PI/180);
	long long  id = blockIdx.x*blockDim.x + threadIdx.x;
	int j = id/nx, i = id%nx;
	float y = j-ny/2, x = i-nx/2;
	if(i>=nx || j>=ny) return;
	
	//Res of rotation from (x,y) 
	float res = 0;

	//(x,y) rotate e with (nx/2,ny/2) (clockwise) 
	float x2 = (cose*x+sine*y)+nx/2;
	float y2 = (-sine*x+cose*y)+ny/2;

	//Ouf of boundary after rotation
	if (x2<0||x2>nx-1.0||y2<0||y2>ny-1.0) res=0;
	else
	{
		int ii,jj;
		int k0,k1,k2,k3;
		float t,u,p0,p1,p2,p3;
		ii=floor(x2);
		jj=floor(y2);
		k0=ii+jj*nx;
		k1=k0+1;
		k2=k0+nx+1;
		k3=k0+nx;

		//handle situation when ii,jj are out of boundary
		if (ii==nx-1) { k1--; k2--; }
		if (jj==ny-1) { k2-=nx; k3-=nx; }
		t=(x2-(float)ii);
		u=(y2-(float)jj);
		float tt=1.0-t;
		float uu=1.0-u;

		//bilinear interpolation of raw data (i,j)(i+1,j)(i,j+1)(i+1,j+1)
		p0=d_image[k0]*tt*uu;
		p1=d_image[k1]*t*uu;
		p3=d_image[k3]*tt*u;
		p2=d_image[k2]*t*u;
		res=p0+p1+p2+p3;

	}

	// res <=> data[i+j*nx] after rotation
	d_rotated_image[id] = res;

}

__global__ void split_IMG(float *Ori,hipfftComplex *IMG, int nx,int ny,int l,int bx,int overlap)
{
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	int image_size = l*l;
	int image_id = i/image_size;
    int local_id = i % image_size;
    int x = local_id % l;
    int y = local_id / l;

	int tmp = l - overlap;

	int area_x_id = image_id%bx;
	int area_y_id = image_id/bx;
	int ori_x = area_x_id*tmp + x;
	int ori_y = area_y_id*tmp + y;

	if(ori_x>=nx || ori_y>=ny) return;
	IMG[i].x = Ori[ori_x + ori_y*nx];
}

//Tl = template(template has been predefined by C++)
__global__ void compute_corner_CCG(hipfftComplex *CCG, hipfftComplex *Tl, hipfftComplex *IMG, int l, int block_id)
{
	//On this function,block means subimage splitted from IMG, not block ON GPU
	long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	//Area of rectangle, l^2
	int l2 = l*l;

	//Local id corresponding to splitted IMG 
	int local_id = i%l2;
	int local_x = local_id%l;
	int local_y = local_id/l;

	int off = block_id * l2;

	//Global ID in IMG
	int j = local_x + local_y*l + off;

	//CCG[i] = IMG'[i]*template[i]
	// ' means conjugate
	CCG[i].x = (IMG[j].x*Tl[i].x+IMG[j].y*Tl[i].y);
	CCG[i].y = (IMG[j].y*Tl[i].x-IMG[j].x*Tl[i].y);

	//Move center to around
	int of = (l/2)%2,st;
	if( of == local_y%2) st = 1; else st = 0;
	if( (local_x-st)%2 == 0 ) 
	{
		CCG[i].x *= -1;
		CCG[i].y *= -1;
	}


}

//"MAX" reduction for *odata : return max{odata[i]},i
//"SUM" reduction for *odata : return sum{odata[i]},sum{odata[i]^2}
__global__ void get_peak_and_SUM(hipfftComplex *odata,float *res,int l,float d_m,int x_bound,int y_bound)
{
	extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    int tid = threadIdx.x;
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	int image_size = l*l;
    int local_id = i % image_size;
    int x = local_id % l;
    int y = local_id / l;

	sdata[tid] = odata[i].x;
	if(x>=x_bound || y>=y_bound ||x<d_m/4 || x>l-d_m/4 || y<d_m/4 || y>l-d_m/4 ) sdata[tid] = 0;
	sdata[tid+blockDim.x] = local_id;
	sdata[tid+2*blockDim.x] = odata[i].x;
	sdata[tid+3*blockDim.x] = odata[i].x*odata[i].x;
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if(tid<s)
		{
			//find max
			if(sdata[tid+s]>sdata[tid]){
				sdata[tid] = sdata[tid+s];
				sdata[tid+blockDim.x] = sdata[tid+blockDim.x+s];
			}
			//sum of data[i] & data[i]^2
			sdata[tid+2*blockDim.x] += sdata[tid+2*blockDim.x + s]; 
			sdata[tid+3*blockDim.x] += sdata[tid+3*blockDim.x + s];
		}
		__syncthreads();
	}
	if(tid==0){
		res[blockIdx.x*4] = sdata[0];
		res[blockIdx.x*4+1] = sdata[blockDim.x];
		res[blockIdx.x*4+2] = sdata[2*blockDim.x];
		res[blockIdx.x*4+3] = sdata[3*blockDim.x];
	}

}

// CUFFT will enlarge VALUE to N times. Restore it
__global__ void scale(hipfftComplex *data,int l2)
{
	long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	data[i].x /= l2;
	data[i].y /= l2;
}

__global__ void clear_image(hipfftComplex *data)
{
	long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	data[i].x = 0;
	data[i].y = 0;
}

__global__ void Complex2float(float *f, hipfftComplex *c, int nx, int ny)
{
	long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= nx*ny) return;
	f[i] = c[i].x;
}

__global__ void float2Complex(hipfftComplex *c, float *f, int nx, int ny)
{
	long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= nx*ny) return;
	c[i].x = f[i] ;
	c[i].y = 0 ;
}

__global__ void do_phase_flip(hipfftComplex *filter, Parameters para, int nx, int ny)
{
    long long  i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= nx*ny) return;
    int x = i % nx;
    int y = i / nx;
	float v=CTF_AST(x,(y+ny/2)%ny,nx,ny,para.apix,para.ds,para.dfu,para.dfv,para.dfdiff,para.dfang,para.lambda,para.cs,para.ampconst,0);

	filter[i].x *= v;
	filter[i].y *= v;

}

void cudaMemoryTest()
{
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    CUDA_CALL(hipMalloc((int**)&d_a, bytes));

    memset(h_a, 0, bytes);
    CUDA_CALL(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
    printf("Test finished.\n");
}