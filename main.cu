#include "hip/hip_runtime.h"
#include "EMReader/emdata_.h"
#include "EMReader/util_func.h"
#include "EMReader/DataReader.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "GPU_func.cuh"
#include <malloc.h>
#include <cmath>
#include <iostream>

using namespace std;

void readEMData(Parameters *para, EulerData *euler)
{
    readEulerData(para->eulerf,euler);
	readSNRWeight(para->snr,&para->a,&para->b,&para->b2,&para->bfactor,&para->bfactor2,&para->bfactor3);
}

//add prefix of inlst to t. 
//.exp  inlst:../test.lsh  t:a.mrc => t:../a.mrc
void addPrefix(char *inlst, char *t)
{
    int l = strlen(inlst);
    char prefix[l+50];
    memset(prefix, 0, (l+50)*sizeof(char));
    int ed = -1;
    for(int i = l-1;i>=0;i--) if(inlst[i] == '/') {ed = i; break;}
    strncpy(prefix,inlst,ed+1);
    strcat(prefix,t);
    strcpy(t,prefix);
#ifdef DEBUG
    printf("--raw_image        %s\n",t);
#endif
}

void readRawImage(emdata *i2d_emdata,Parameters *para,int n, int *nn, char* t)
{
    vector<string> pairs;
    int return_code = readInLst_and_consturctPairs(para->inlst,t,&pairs,nn,n);
    if( return_code < 0) printf("Error %d occured in readRawImage!\n",return_code);

    //add prefix of inlst to t. 
    //.exp  inlst:../test.lsh  t:a.mrc => t:../a.mrc
    addPrefix(para->inlst,t);
    i2d_emdata->readImage(t,*nn);
    parsePairs(pairs, &para->defocus, &para->dfdiff, &para->dfang);
    
    para->dfu=para->defocus+para->dfdiff; //defocus is minus, so abs(dfu) < abs(dfv)
    para->dfv=para->defocus-para->dfdiff;
    para->lambda=12.2639/sqrt(para->energy*1000.0+0.97845*para->energy*para->energy);
    para->ds=1/(para->apix * para->padding_size);
}


//read template(float *)
//covert template from float* to cufftcomplex *
void readAndPaddingTemplate(Parameters *para,hipfftComplex *h_templates,int N, double *sigmas)
{
    int padded_template_size = para->padding_size * para->padding_size;
    emdata *tp = new emdata();
    for(int J=0;J<N;J++)
    {
        tp->readImage(para->temp2d,J);
        float *data = tp->getData();
        if(para->padding_size < tp->header.nx || para->padding_size < tp->header.ny)
        {
            printf("Padded size is smaller than template.nx /ny\n");
            exit(-1) ;
        }
        int sx = (para->padding_size - tp->header.nx)/2;
        int sy = (para->padding_size - tp->header.ny)/2;
        for(int j=0;j<tp->header.ny;j++)
            for(int i=0;i<tp->header.nx;i++)
            {
                long long index = padded_template_size*J + (sy+j)*para->padding_size + (sx+i);
                h_templates[index].x = data[i+j*tp->header.nx];
            }
    }
	para->template_x = tp->header.nx;
    para->template_y = tp->header.ny;
    para->template_z = tp->header.nz;
    //para->overlap = para->template_x*0.13+1;
    //free heap memory
    if(tp!=NULL) delete tp;
    free(tp);
}

void cudaAllocTemplateMem(int N, int nx, int ny, Parameters *para,float **h_reduction_buf,float **d_reduction_buf,float **d_means,double **sigmas,double **d_sigmas,hipfftComplex **h_templates,
    hipfftComplex **d_templates,hipfftComplex **CCG,hipStream_t *stream,float **ra, float **rb, hipfftHandle *plan_for_temp)
{
    int tmp = (para->padding_size - para->overlap);
    //num of blocks in x,y axis
    int block_x = (nx-para->overlap) / tmp;
    if((nx-para->overlap) % tmp > 0 ) block_x ++; 
    int block_y = (ny-para->overlap) / tmp;
    if((nx-para->overlap) % tmp > 0 ) block_y ++; 
    para->block_x = block_x;
    para->block_y = block_y;
    
    //N = max{num of tmplates ,  num of subimgs }
    N = max(N,block_x*block_y);

    //Number of Pixels for each padded template
    long long padded_template_size = para->padding_size*para->padding_size;

    //All padded templates (complex) At CPU,GPU
    *h_templates = (hipfftComplex *)malloc(sizeof(hipfftComplex)*padded_template_size*N);
    memset(*h_templates,0,sizeof(hipfftComplex)*padded_template_size*N);
    CUDA_CALL(  hipMalloc(d_templates,sizeof(hipfftComplex)*padded_template_size*N)  );
    
    //Memory alloc for CCG
    CUDA_CALL(  hipMalloc(CCG,sizeof(hipfftComplex)*padded_template_size*N)  );

    //Store sigma value for all templates
    *sigmas = (double *)malloc(sizeof(double)*N);
    CUDA_CALL(  hipMalloc(d_sigmas,sizeof(double)*N)  );
    
    //Cuda Stream
	hipStreamCreate(stream);

    //Temp buffer for whiten
    CUDA_CALL(  hipMalloc(ra,N*(RA_SIZE)*sizeof(float))  );
    CUDA_CALL(  hipMalloc(rb,N*(RA_SIZE)*sizeof(float))  );

    //Buffer for reduction
    long long buf_size = max((long long)RA_SIZE*RA_SIZE/BLOCK_SIZE, 4*padded_template_size*N/BLOCK_SIZE);
    *h_reduction_buf = (float *)malloc(buf_size*sizeof(float));
    CUDA_CALL(  hipMalloc(d_reduction_buf,buf_size*sizeof(float))  );

    //Store mean for every template
    CUDA_CALL(  hipMalloc(d_means,sizeof(float)*N)  );

    /*
	hipfftMakePlanMany(hipfftHandle plan, int rank, int *n, int *inembed,
	int istride, int idist, int *onembed, int ostride,
	int odist, hipfftType type, int batch, size_t *workSize);
	 */
	const int rank = 2;//维数
    int n[rank] = { para->padding_size, para->padding_size };//n*m
    int *inembed = n;//输入的数组size
    int istride = 1;//数组内数据连续，为1
    int idist = n[0] * n[1];//1个数组的内存大小
    int *onembed = n;//输出是一个数组的size
    int ostride = 1;//每点DFT后数据连续则为1
    int odist = n[0] * n[1];//输出第一个数组与第二个数组的距离，即两个数组的首元素的距离
    int batch = N;//批量处理的批数
    //采用hipfftPlanMany方法
    
    //FFT handler for all templates
    CUFFT_CALL(hipfftPlanMany(plan_for_temp, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));//针对多信号同时进行FFT
    //Binding stream to plan
    CUFFT_CALL(hipfftSetStream(*plan_for_temp, *stream));

}

void edgeNormalize(int N,double *sigmas,double *d_sigmas,hipfftComplex *mask,hipfftComplex *h_templates,hipfftComplex *d_templates,float *h_buf,float *d_buf,
    float *N_buffer,Parameters para,hipStream_t *stream)
{
    int l = para.padding_size;
    long long padded_template_size = l*l;
    int block_num = padded_template_size*N/BLOCK_SIZE;

    float r = l/(float)2-2;
    float up_bound = 0, low_bound = 0;
    if(r > 1)
    {
        up_bound = (r+1)*(r+1);
        low_bound = (r-1)*(r-1);
    }

    //Clear to {0}
    clear_image<<<block_num,BLOCK_SIZE,0,*stream>>>(mask);

    //em = cirMean()
    //generate Mask, count numbor of no-zero digits
    generate_mask<<<block_num,BLOCK_SIZE,BLOCK_SIZE*sizeof(float),*stream>>>(l,mask,r,d_buf,up_bound,low_bound);
    CUDA_CHECK();

    float N_buffer_host[2*N];
    CUDA_CALL(hipMemcpyAsync(h_buf, d_buf,sizeof(float)*padded_template_size*N/BLOCK_SIZE, hipMemcpyDeviceToHost, *stream));
    memset(N_buffer_host,0,N*sizeof(float));
    for(int k=0;k<padded_template_size*N/BLOCK_SIZE;k++)
    {
        int id = k/(padded_template_size/BLOCK_SIZE);
        N_buffer_host[id] += h_buf[k];
        //Number for every no-zero digits
    }

    //Calculate dot of mask and all templates
    CUDA_CALL(  hipMemcpyAsync(N_buffer, N_buffer_host, sizeof(float)*N, hipMemcpyHostToDevice, *stream)  );
    multiCount_dot<<<block_num,BLOCK_SIZE,BLOCK_SIZE*sizeof(float),*stream>>>(l,mask,d_templates,N_buffer,d_buf);
    CUDA_CHECK();
    CUDA_CALL(hipMemcpyAsync(h_buf, d_buf,sizeof(float)*padded_template_size*N/BLOCK_SIZE, hipMemcpyDeviceToHost, *stream));
    memset(N_buffer_host,0,N*sizeof(float));
    for(int k=0;k<padded_template_size*N/BLOCK_SIZE;k++)
    {
        int id = k/(padded_template_size/BLOCK_SIZE);
        N_buffer_host[id] += h_buf[k];
        //Dot result:em
    }
    
    CUDA_CALL(  hipMemcpyAsync(d_templates, h_templates, sizeof(hipfftComplex)*padded_template_size*N, hipMemcpyHostToDevice, *stream)  );
    UpdateSigma<<<block_num,BLOCK_SIZE,BLOCK_SIZE*sizeof(float)*2,*stream>>>(d_templates,d_buf);
    CUDA_CHECK();
    CUDA_CALL(  hipMemcpyAsync(h_buf, d_buf,2*sizeof(float)*padded_template_size*N/BLOCK_SIZE, hipMemcpyDeviceToHost, *stream));
    
    //put em on GPU
    CUDA_CALL(  hipMemcpyAsync(N_buffer, N_buffer_host, sizeof(float)*N, hipMemcpyHostToDevice, *stream)  );

    memset(N_buffer_host,0,2*N*sizeof(float));
    for(int k=0;k<padded_template_size*N/BLOCK_SIZE;k++)
    {
        int id = k/(padded_template_size/BLOCK_SIZE);
        N_buffer_host[2*id] += h_buf[2*k];
        //sum of value
        N_buffer_host[2*id+1] += h_buf[2*k+1];
        //sum of value^2
    }

    for(int i=0;i<N;i++) {
        double mean = N_buffer_host[2*i] / (double)(l*l);
        sigmas[i] = sqrt(N_buffer_host[2*i+1] / (double)(l*l) - mean*mean);
        if(sigmas[i]<0 || !finite(sigmas[i])) sigmas[i] = 0 ;
    }

    //data[i]=(data[i]-em)/s;
    CUDA_CALL(  hipMemcpyAsync(d_sigmas, sigmas, sizeof(double)*N, hipMemcpyHostToDevice, *stream)  );
    scale_each<<<block_num,BLOCK_SIZE,0,*stream>>>(l,d_templates,N_buffer,d_sigmas);
    CUDA_CHECK();
}


void handleTemplate(int N, float *ra, float *rb,float *h_buf,float *d_buf,float *d_means,
    hipfftComplex *h_templates,hipfftComplex *d_templates,Parameters *para,hipStream_t *stream, hipfftHandle *plan_for_temp)
{

//***************************************************************
// 1. apply whitening filter
// 2. apply mask
// 3. low pass filter and apply weighting function
// 4. normalize
//***************************************************************

    long long padded_template_size = para->padding_size*para->padding_size;
    int block_num = padded_template_size*N/BLOCK_SIZE;

// **************************************************************
// apply whitening filter and do ift
// input: Padded IMAGE (Real SPACE)
// output: IMAGE_whiten (Fourier SPACE in RI)
// **************************************************************
//    CUDA_CALL(    ); => CUDA Exeception handler

    // Inplace FFT
    CUFFT_CALL(  hipfftExecC2C(*plan_for_temp, d_templates, d_templates, HIPFFT_FORWARD)  );
    // CUFFT will enlarge VALUE to N times. Restore it
    scale<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,padded_template_size);
    CUDA_CHECK();

    //Whiten at fourier space
    //contain ri2ap
    SQRSum_by_circle<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,ra,rb,para->padding_size,para->padding_size);
    CUDA_CHECK();

    //contain ap2ri
    whiten_Tmp<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,ra,rb,para->padding_size);
    CUDA_CHECK();

// **************************************************************
// apply mask 
// input: whiten_IMAGE (Fourier SPACE in RI)
// output: masked_whiten_IMAGE (Fourier SPACE in RI)
// **************************************************************
    CUFFT_CALL(  hipfftExecC2C(*plan_for_temp, d_templates, d_templates, HIPFFT_BACKWARD)  );
    apply_mask<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,para->d_m,para->edge_half_width,para->padding_size);
    CUDA_CHECK();
    CUFFT_CALL(  hipfftExecC2C(*plan_for_temp, d_templates, d_templates, HIPFFT_FORWARD)  );
    // CUFFT will enlarge VALUE to N times. Restore it
    scale<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,padded_template_size);
    CUDA_CHECK();

// **************************************************************
// 1. lowpass
// 2. apply weighting function
// 3. normlize
// input: masked_whiten_IMAGE (Fourier SPACE in RI) 
// output: PROCESSED_IMAGE (Fourier SPACE in AP)
// **************************************************************
    //contain ri2ap
    apply_weighting_function<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,*para);
    CUDA_CHECK();

    compute_area_sum_ofSQR<<<block_num,BLOCK_SIZE,2*BLOCK_SIZE*sizeof(float),*stream>>>(d_templates,d_buf,para->padding_size,para->padding_size);
    CUDA_CHECK();
    CUDA_CALL(hipMemcpyAsync(h_buf, d_buf,2*sizeof(float)*padded_template_size*N/BLOCK_SIZE, hipMemcpyDeviceToHost, *stream));
    hipStreamSynchronize(*stream);
    //After Reduction -> compute mean for each image
    float infile_mean[N],counts[N];
    memset(infile_mean,0,N*sizeof(float));
    memset(counts,0,N*sizeof(float));
    for(int k=0;k<padded_template_size*N/BLOCK_SIZE;k++)
    {
        int id = k/(padded_template_size/BLOCK_SIZE);
        infile_mean[id] += h_buf[2*k];
        counts[id] += h_buf[2*k+1];
    }
    for(int k=0;k<N;k++) infile_mean[k] = sqrtf(infile_mean[k]/(counts[k]*counts[k]));
    //Do Normalization with computed infile_mean[]
    CUDA_CALL(  hipMemcpyAsync(d_means, infile_mean, sizeof(float)*N, hipMemcpyHostToDevice, *stream)  );
    //Contain ap2ri
    normalize<<<block_num,BLOCK_SIZE,0,*stream>>>(d_templates,para->padding_size,para->padding_size,d_means);
    CUDA_CHECK();

}

void cudaAllocImageMem(float **d_image,hipfftComplex **d_rotated_image,hipfftComplex **rotated_splitted_image,hipStream_t *stream,
    hipfftHandle *plan_for_image,hipfftHandle *plan_for_whole_IMG,int nx,int ny,int N,Parameters *para)
{
    //X Y Size for padded IMG
    int ix = para->block_x*para->padding_size;
    int iy = para->block_y*para->padding_size;

    CUDA_CALL(  hipMalloc(d_image,nx*ny*sizeof(float))  );
    CUDA_CALL(  hipMalloc(d_rotated_image,ix*iy*sizeof(hipfftComplex))  );
    CUDA_CALL(  hipMalloc(rotated_splitted_image,ix*iy*sizeof(hipfftComplex))  );

    /*
	hipfftMakePlanMany(hipfftHandle plan, int rank, int *n, int *inembed,
	int istride, int idist, int *onembed, int ostride,
	int odist, hipfftType type, int batch, size_t *workSize);
	 */
	const int rank = 2;//维数
    int n[rank] = { para->padding_size, para->padding_size };//n*m
    int *inembed = n;//输入的数组size
    int istride = 1;//数组内数据连续，为1
    int idist = n[0] * n[1];//1个数组的内存大小
    int *onembed = n;//输出是一个数组的size
    int ostride = 1;//每点DFT后数据连续则为1
    int odist = n[0] * n[1];//输出第一个数组与第二个数组的距离，即两个数组的首元素的距离
    int batch = para->block_x*para->block_y;//批量处理的批数
    //采用hipfftPlanMany方法
    
    //FFT handler for all sub images
    CUFFT_CALL(hipfftPlanMany(plan_for_image, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));//针对多信号同时进行FFT
    //Binding stream to plan
    CUFFT_CALL(hipfftSetStream(*plan_for_image, *stream));

    int n2[rank] = { nx, ny };//n*m
    inembed = n2;//输入的数组size
    istride = 1;//数组内数据连续，为1
    idist = n2[0] * n2[1];//1个数组的内存大小
    onembed = n2;//输出是一个数组的size
    ostride = 1;//每点DFT后数据连续则为1
    odist = n2[0] * n2[1];//输出第一个数组与第二个数组的距离，即两个数组的首元素的距离
    batch = 1;//批量处理的批数
    //FFT handler for single whole images
    CUFFT_CALL(hipfftPlanMany(plan_for_whole_IMG, rank, n2, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));//针对多信号同时进行FFT
    //Binding stream to plan
    CUFFT_CALL(hipfftSetStream(*plan_for_whole_IMG, *stream));
}  

void init_d_image(Parameters para,hipfftComplex *filter,float *d_image, float*ra, float *rb, emdata *image, int nx, int ny, hipStream_t *stream,hipfftHandle *plan_for_whole_IMG)
{
    //Translate origin of Image to (0,0)
    image->rotate(0);
    //Put Image on GPU
    CUDA_CALL(  hipMemcpyAsync(d_image, image->getData(), sizeof(float)*nx*ny, hipMemcpyHostToDevice, *stream)  );
    if(para.phase_flip == 1)
    {
        int block_num = nx*ny/BLOCK_SIZE+1;
        float2Complex<<<block_num,BLOCK_SIZE,0,*stream>>>(filter,d_image,nx,ny);
        //fft inplace
        CUFFT_CALL(hipfftExecC2C(*plan_for_whole_IMG, filter, filter, HIPFFT_FORWARD));
        scale<<<block_num,BLOCK_SIZE,0,*stream>>>(filter,nx*ny);
        CUDA_CHECK();

        //phase flip
        do_phase_flip<<<block_num,BLOCK_SIZE,0,*stream>>>(filter,para,nx,ny);
        CUDA_CHECK();

        //Whiten at fourier space
        clear_float<<<RA_SIZE/BLOCK_SIZE+1,BLOCK_SIZE,0,*stream>>>(ra);
        clear_float<<<RA_SIZE/BLOCK_SIZE+1,BLOCK_SIZE,0,*stream>>>(rb);
        //contain ri2ap
        SQRSum_by_circle<<<block_num,BLOCK_SIZE,0,*stream>>>(filter,ra,rb,nx,ny,1);
        CUDA_CHECK();

        // 1. whiten
        // 2. low pass
        // 3. weight
        // 4. ap2ri
        whiten_filetr_weight_Img<<<block_num,BLOCK_SIZE,0,*stream>>>(filter,ra,rb,nx,ny,para);
        CUDA_CHECK();

        //ifft inplace
        CUFFT_CALL(hipfftExecC2C(*plan_for_whole_IMG, filter, filter, HIPFFT_BACKWARD));
        Complex2float<<<block_num,BLOCK_SIZE,0,*stream>>>(d_image,filter,nx,ny);

    }

}

void split_normalize_image(float *d_image,hipfftComplex *d_rotated_image,float *h_buf,float *d_buf,float *d_means, Parameters para, hipStream_t *stream, int nx, int ny, hipfftHandle *image_plan)
{
    int l = para.padding_size;
    // Init d_rotated_imge to all {0}
    int ix = para.block_x;
    int iy = para.block_y;
    int blockIMG_num = ix*iy*l*l / BLOCK_SIZE;
    clear_image<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(d_rotated_image);
    CUDA_CHECK();
    // split Image into blocks with overlap  
    split_IMG<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(d_image,d_rotated_image,nx,ny,para.padding_size,para.block_x,para.overlap);	
    CUDA_CHECK();

    // do normalize to all subIMGs
    if(para.phase_flip == 1)
    {
        //Inplace FFT
        CUFFT_CALL(  hipfftExecC2C(*image_plan, d_rotated_image, d_rotated_image, HIPFFT_FORWARD)  );
        //Scale IMG to normel size
        scale<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(d_rotated_image,l*l);
        CUDA_CHECK();

        ri2ap<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(d_rotated_image);

        compute_area_sum_ofSQR<<<blockIMG_num,BLOCK_SIZE,2*BLOCK_SIZE*sizeof(float),*stream>>>(d_rotated_image,d_buf,l,l);
        CUDA_CHECK();
        CUDA_CALL(hipMemcpyAsync(h_buf, d_buf,2*sizeof(float)*blockIMG_num, hipMemcpyDeviceToHost, *stream));
        hipStreamSynchronize(*stream);

        int N_IMG = para.block_x*para.block_y;
        //After Reduction -> compute mean for each image
        float infile_mean[N_IMG],counts[N_IMG];
        memset(infile_mean,0,N_IMG*sizeof(float));
        memset(counts,0,N_IMG*sizeof(float));
        for(int k=0;k<blockIMG_num;k++)
        {
            int id = k/( (l*l)/BLOCK_SIZE );
            infile_mean[id] += h_buf[2*k];
            counts[id] += h_buf[2*k+1];
        }
        for(int k=0;k<N_IMG;k++) infile_mean[k] = sqrtf(infile_mean[k]/(counts[k]*counts[k]));
        
        //Do Normalization with computed infile_mean[]
        CUDA_CALL(  hipMemcpyAsync(d_means, infile_mean, sizeof(float)*N_IMG, hipMemcpyHostToDevice, *stream)  );
        //Contain ap2ri
        normalize<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(d_rotated_image,l,l,d_means);
        CUDA_CHECK();

        //Inplace IFT
        CUFFT_CALL(  hipfftExecC2C(*image_plan, d_rotated_image, d_rotated_image, HIPFFT_BACKWARD)  );
    }
    
}

void rotateImage(hipfftComplex *splitted_image,hipfftComplex *rotated_splitted_image,Parameters para, float e, hipStream_t *stream)
{
    // Init d_rotated_imge to all {0}
    int ix = para.block_x;
    int iy = para.block_y;
    int blockIMG_num = ix*iy*para.padding_size*para.padding_size / BLOCK_SIZE;
    // rotate subIMG with angle "e"
    rotate_subIMG<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(splitted_image,rotated_splitted_image,e,para.padding_size);
    CUDA_CHECK();

}

void pickPartcles(hipfftComplex *CCG,hipfftComplex *d_templates,hipfftComplex *rotated_splitted_image,float *h_buf,float *d_buf, Parameters para, 
    hipfftHandle *template_plan, hipfftHandle *image_plan,hipStream_t *stream,int N,float *scores, int nx, int ny, float euler3)
{           
    int l = para.padding_size;
    long long padded_template_size = l*l;
    int blockGPU_num = padded_template_size*N/BLOCK_SIZE;
    int blockIMG_num = padded_template_size*para.block_x*para.block_y/BLOCK_SIZE;

    //peak,sum of data[i],sum of data[i]^2
    float peaks[N],pos[N],sums[N],sum2s[N];

    //find MAX score need initialize
    memset(scores,0,sizeof(float)*3*N);

    //Inplace FFT
    CUFFT_CALL(  hipfftExecC2C(*image_plan, rotated_splitted_image, rotated_splitted_image, HIPFFT_FORWARD)  );

    //Scale IMG to normel size
    scale<<<blockIMG_num,BLOCK_SIZE,0,*stream>>>(rotated_splitted_image,padded_template_size);
    CUDA_CHECK();

    //compute score for each block
    for(int j=0;j<para.block_y;j++)
    {
        for(int i=0;i<para.block_x;i++)
        {
            //find peak need initialize
            memset(peaks,0,sizeof(float)*N);
            memset(pos,0,sizeof(float)*N);
            memset(sums,0,sizeof(float)*N);
            memset(sum2s,0,sizeof(float)*N);
            //compute CCG
            compute_corner_CCG<<<blockGPU_num,BLOCK_SIZE,0,*stream>>>(CCG,d_templates,rotated_splitted_image,l,i+j*para.block_x);
            CUDA_CHECK();
            //Inplace IFT
            CUFFT_CALL(  hipfftExecC2C(*template_plan, CCG, CCG, HIPFFT_BACKWARD)  );
            //Ingore padded 0 at raw IMG
            int x_bound = nx - i*(l-para.overlap);
            int y_bound = ny - j*(l-para.overlap);
            //find peak(position) and get sum of data,data^2
            get_peak_and_SUM<<<blockGPU_num,BLOCK_SIZE,4*BLOCK_SIZE*sizeof(float),*stream>>>(CCG,d_buf,l,para.d_m,x_bound,y_bound);
            CUDA_CHECK();
            CUDA_CALL(hipMemcpyAsync(h_buf, d_buf, 4*sizeof(float)*padded_template_size*N/BLOCK_SIZE, hipMemcpyDeviceToHost, *stream));
            hipStreamSynchronize(*stream);

            //After Reduction -> compute mean for each image
            for(int k=0;k<(padded_template_size*N)/BLOCK_SIZE;k++)
            {
                int id = k/(padded_template_size/BLOCK_SIZE);
                if(peaks[id] < h_buf[4*k])
                {
                    peaks[id] = h_buf[4*k];
                    pos[id] = h_buf[4*k+1];
                }
                sums[id] += h_buf[4*k+2];
                sum2s[id] += h_buf[4*k+3];
            }

            //Update global score with local-block score for each template
            for(int J=0;J<N;J++) 
            {
                float ra = sums[J]-peaks[J];
                float rb = sum2s[J]-peaks[J]*peaks[J];
                float rc = padded_template_size - 1;
                float sd = sqrt(rb/rc - (ra/rc)*(ra/rc));
                float score;
                if(sd == 0) score = 0;
                else score = peaks[J]/sqrt(rb/rc - (ra/rc)*(ra/rc));
                
                int cx = (int)pos[J]%l;
                int cy = (int)pos[J]/l;

                //Rotate (cx,cy) to its soriginal angle
                float centerx =  i*(l-para.overlap) + (cx-l/2)*cos(euler3*PI/180)+(cy-l/2)*sin(euler3*PI/180)+l/2; // centerx
                float centery =  j*(l-para.overlap) + (cy-l/2)*cos(euler3*PI/180)-(cx-l/2)*sin(euler3*PI/180)+l/2; // centery
                //if(J==0) printf("%d %d %f %f %f %d %d =>(%d %d)\n",i,j,centerx,centery,score,cx,cy,i*(l-para.overlap),j*(l-para.overlap));

                if(scores[3*J] < score)
                {                    
                    //float Ny = para.d_m;
                    //if(cy-Ny/3>=0 && cx-Ny/3>=0 && cy+Ny/3<=ny && cx+Ny/3<=nx)
                    if(centerx>=0 && centerx<nx &&centery>=0 &&centery<ny)
                    {
                        scores[3*J] = score;
                        scores[3*J+1] = centerx;
                        scores[3*J+2] = centery;
                        //if(centerx <Ny/3 || centery<Ny/3 || centerx>(ny-Ny/3) || centery>(ny-Ny/3)) scores[3*k]=0;
                    }
                }

            }
        }
    }
}

void writeScoreToDisk(int N_tmp,float *scores,Parameters para,EulerData euler,FILE *fp, int nn, char *t, int nx, int ny, float euler3)
{
    for(int J=0;J<N_tmp;J++)
    {
        float score = scores[3*J];
        float centerx = scores[3*J+1];
        float centery = scores[3*J+2];

        if(score > para.thres)
        {
            fprintf(fp, "%d\t%s\tdefocus=%f\tdfdiff=%f\tdfang=%f\teuler=%f,%f,%f\tcenter=%f,%f\tscore=%f\n",
                    nn,t,(-1)*para.defocus,para.dfdiff,para.dfang,euler.euler1[J],euler.euler2[J],euler3,centerx,centery,score
            );
        }
    }
}

int main(int argc, char *argv[])
{
    //Timer
    time_t first, second;  
    first=time(NULL);

    //euler.length, number of template
    int N_tmp;

    //Print Help Message if no para input
    if(argc==1){
		printHelpMsg(); 
		return 0;
	}

    //Store all parameters(some from input, some from computation)
    Parameters para;
    //Stroe Euler data
    EulerData euler;
    
    readParameters(argc,argv,&para);
#ifdef DEBUG
    para.printAllPara();
#endif
    readEMData(&para,&euler);
    N_tmp = euler.length; 

    //IMG id
    int nn=0;
    //IMG filename
    char t[MAXPATHLEN+1];
    //Used to write res
    FILE *fp=fopen(para.outlst,"wb");
    // Set GPU device ID
    CUDA_CALL(  hipSetDevice(para.device_id)  );

//*************************************************
// Process All Templates
// 1. Alloc memory
// 2. Read Template
// 3. Preprocess Template
//*************************************************
    //Store templates at CPU/GPU
    hipfftComplex *h_templates,*d_templates;
    //Store res = F(template)F'(Sliptted_padded_IMG)
    hipfftComplex *CCG;
    //Temp buffer for whiten
    float *ra,*rb;
    //Buffer for reduction
    float *h_reduction_buf,*d_reduction_buf;
    //Store mean for every template at GPU
    float *d_means;
    //Cuda Stream
    hipStream_t stream;
    //cufft handler for template FFT
    hipfftHandle plan_for_temp;
    //Store sigma value for all templates
    double *sigmas;
    double *d_sigmas;

    // READ FIRST IMG TO GET nx & ny
    //Image to be searched 
    emdata *image = new emdata();
    //Read Image
    readRawImage(image,&para,para.first,&nn,t);
    //size of first IMG
    int nx = image->header.nx;
    int ny = image->header.ny;
#ifdef DEBUG
        printf("IMG size:           %d %d\n",nx,ny);
        printf("Number of template: %d\n",N_tmp);
#endif
    cudaAllocTemplateMem(N_tmp,nx,ny,&para,&h_reduction_buf,&d_reduction_buf,&d_means,&sigmas,&d_sigmas,
        &h_templates,&d_templates,&CCG,&stream,&ra,&rb,&plan_for_temp);
    //free heap memory
    if(image!=NULL) delete image;

    //Loop for all Images. (Last - First) normally is 1;
    for(int n=para.first;n<para.last;n++)
	{
        if(n != para.first)
        {
            //Image to be searched 
            image = new emdata();
            //Read Image
            readRawImage(image,&para,n,&nn,t);
        }
        //Reading Template
        readAndPaddingTemplate(&para,h_templates,N_tmp,sigmas);

        //Process edge normlization
        //To avoid alloc memory,use CCG as tempary buf
        edgeNormalize(N_tmp,sigmas,d_sigmas,CCG,h_templates,d_templates,h_reduction_buf,d_reduction_buf,d_means,para,&stream);

        //whiten, apply mask, appy weighting ..
        handleTemplate(N_tmp,ra,rb,h_reduction_buf,d_reduction_buf,d_means,
            h_templates,d_templates,&para,&stream,&plan_for_temp);

//*************************************************
// Process Image to be searched 
// 0. Read
// 1. Rotate
// 2. Split
// 3. doFFT
//*************************************************   
        float *d_image;
        hipfftComplex *d_rotated_image;
        hipfftComplex *rotated_splitted_image;
        //cufft handler for sub-IMGs and whole-IMG FFT
        hipfftHandle plan_for_image,plan_for_whole_IMG;
        
        cudaAllocImageMem(&d_image,&d_rotated_image,&rotated_splitted_image,&stream,&plan_for_image,&plan_for_whole_IMG,nx,ny,N_tmp,&para);
        // 1.Put Image on GPU 2.phaseflip
        init_d_image(para,rotated_splitted_image,d_image,ra,rb,image,nx,ny,&stream,&plan_for_whole_IMG);
        // split Image into blocks with overlap
        split_normalize_image(d_image,d_rotated_image,h_reduction_buf,d_reduction_buf,d_means,para,&stream,nx,ny,&plan_for_image);

        //Scores : [1-N]->socre  [N+1-2N]->cx+cy*padding_size
        float *scores = new float[N_tmp*3];    
        for(float euler3=0.0;euler3<360.0;euler3+=para.phi_step)
        {	 
#ifdef DEBUG
            printf("Now euler3 => %f / 360.0\n",euler3);
#endif 
            // rotate Image  
            rotateImage(d_rotated_image,rotated_splitted_image,para,euler3,&stream);

            //***************************************************
            //Pick particles from IMG with template
            //1. Calculate ccg
            //2. Find peak 
            //3. Calculate variance
            //4. Output score
            //***************************************************

            pickPartcles(CCG,d_templates,rotated_splitted_image,h_reduction_buf,d_reduction_buf,para,&plan_for_temp,&plan_for_image,&stream,N_tmp,scores,nx,ny,euler3);
            hipStreamSynchronize(stream);
            writeScoreToDisk(N_tmp,scores,para,euler,fp,nn,t,nx,ny,euler3);
        }
        hipfftDestroy(plan_for_whole_IMG);
        hipfftDestroy(plan_for_image);
        hipFree(rotated_splitted_image);
        hipFree(d_rotated_image);
        hipFree(d_image);
        delete []scores;
        
        //free heap memory
        if(image!=NULL) delete image;
    }

    hipfftDestroy(plan_for_temp);
    hipStreamDestroy(stream);
    hipFree(d_reduction_buf);
    hipFree(d_templates);
    hipFree(d_sigmas);
    hipFree(d_means);
    hipFree(CCG);
    hipFree(ra);
    hipFree(rb);
    free(h_reduction_buf);
    free(h_templates);
    free(sigmas);
    fclose(fp);

    //Timer
    second=time(NULL);  
    printf("Total consumed time is: %f seconds\n",difftime(second,first)); 

    return 0;
}
